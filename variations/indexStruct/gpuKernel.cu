#include "hip/hip_runtime.h"
#include "gpuKernel.h"
#include "Idx.h"
#include "mms.h"
#include "CuErr.h"
#include <iostream>
__global__ void K_Init(FlowArr flow, FlowArr err, const InputClass input, const int lb)
{
    int i = threadIdx.x + blockIdx.x*blockDim.x - input.nguard;
    int j = threadIdx.y + blockIdx.y*blockDim.y - input.nguard;
#if(IS3D)    
    int k = threadIdx.z + blockIdx.z*blockDim.z - input.nguard;
#else
    int k = 0;
#endif
    double xyz[3];
    double dx[3];
    dx[0] = (input.bounds[1] - input.bounds[0])/input.nxb[0];
    dx[1] = (input.bounds[3] - input.bounds[2])/input.nxb[1];
#if(IS3D)
    dx[2] = (input.bounds[5] - input.bounds[4])/input.nxb[2];
#else
    dx[2] = 0.0;
#endif

    if (i < (input.nxb[0]+input.nguard) && j < (input.nxb[1]+input.nguard) && k < (input.nxb[2]+input.nguard))
    {
        double pres[4];
        double dens[4];
        double uvel[4];
        double vvel[4];
        double wvel[4];
        
        xyz[0] = input.bounds[0] + (i + 0.5)*dx[0];
        xyz[1] = input.bounds[2] + (j + 0.5)*dx[1];
#if(IS3D)
        xyz[2] = input.bounds[4] + (k + 0.5)*dx[2];
#else
        xyz[2] = 0.0;
#endif

        pres_mms(pres, xyz[0], xyz[1], xyz[2]);
        dens_mms(dens, xyz[0], xyz[1], xyz[2]);
        uvel_mms(uvel, xyz[0], xyz[1], xyz[2]);
        vvel_mms(vvel, xyz[0], xyz[1], xyz[2]);
        wvel_mms(wvel, xyz[0], xyz[1], xyz[2]);

        flow(0, i, j, k, lb) = pres[0];
        flow(1, i, j, k, lb) = dens[0];
        flow(2, i, j, k, lb) = uvel[0];
        flow(3, i, j, k, lb) = vvel[0];
#if(IS3D)
        flow(4, i, j, k, lb) = wvel[0];
#endif
        err(0, i, j, k, lb) = 0.0;
        err(1, i, j, k, lb) = 0.0;
        err(2, i, j, k, lb) = 0.0;
        err(3, i, j, k, lb) = 0.0;
#if(IS3D)
        err(4, i, j, k, lb) = 0.0;
#endif
    }
}

void InitGpu(FlowArr& flow, FlowArr& err, const InputClass& input)
{
    //Global memory:            7.907288 GB
    //Shared memory per block:  48.000000 KB
    //Warp size:                32￼
    //Max threads per block:    1024
    //Max thread dimension:     1,024  x  1,024  x  64
    //Max grid size:            2,147,483,647  x  65,535  x  65,535
    //Total constant memory:    64.000000 KB
    
    dim3 blockConf;
    blockConf.x = BLOCK_SIZEX;
    blockConf.y = BLOCK_SIZEY;
#if(IS3D)
    blockConf.z = BLOCK_SIZEZ;
#endif
    dim3 gridConf;
    int numcells[DIM];
    for (int i = 0; i < DIM; i++) {numcells[i] = input.nxb[i] + 2*input.nguard;}
    gridConf.x = (numcells[0] + BLOCK_SIZEX - 1)/BLOCK_SIZEX;
    gridConf.y = (numcells[1] + BLOCK_SIZEY - 1)/BLOCK_SIZEY;
#if(IS3D)
    gridConf.z = (numcells[2] + BLOCK_SIZEZ - 1)/BLOCK_SIZEZ;
#endif

    if (mypenoG==0 && !hasPrintedGp)
    {
        hasPrintedGp = true;
        std::cout << "GP Config:\nblock: " << blockConf.x << " x " << blockConf.y;
        if (IS3D) std::cout << " x " << blockConf.z;
        std::cout << "\ngrid:  " << gridConf.x << " x " << gridConf.y;
        if (IS3D) std::cout << " x " << gridConf.z;
        std::cout << std::endl;
    }
    
    for (int lb = 0; lb < input.lnblocks; lb++)
    {
        K_Init<<<gridConf, blockConf>>>(flow, err, input, lb);
        CuCheck(hipPeekAtLastError());
    }
    CuCheck(hipDeviceSynchronize());
}


#define stencilIdx(v,j) ((v)+(5+DIM)*(j))

#define f_DivSplit(q,j,l,v1)         (0.500*(q[stencilIdx((v1),(j))] + q[stencilIdx((v1),(j)+(l))]))
#define fg_QuadSplit(q,j,l,v1,v2)    (0.250*(q[stencilIdx((v1),(j))] + q[stencilIdx((v1),(j)+(l))])*(q[stencilIdx((v2),(j))] + q[stencilIdx((v2),(j)+(l))]))
#define fg_CubeSplit(q,j,l,v1,v2,v3) (0.125*(q[stencilIdx((v1),(j))] + q[stencilIdx((v1),(j)+(l))])*(q[stencilIdx((v2),(j))] + q[stencilIdx((v2),(j)+(l))])*(q[stencilIdx((v3),(j))] + q[stencilIdx((v3),(j)+(l))]))
#define fg_DivSplit(q,j,l,v1,v2)     (0.500*((q[stencilIdx((v1),(j)+(l))]*q[stencilIdx((v2),(j))]) + (q[stencilIdx((v1),(j))]*q[stencilIdx((v2),(j)+(l))])))

__global__ void K_Conv(FlowArr flow, FlowArr err, const InputClass input, const int lb, const Coef_t center, int stencilWid)
{
    int i = threadIdx.x + blockIdx.x*blockDim.x;
    int j = threadIdx.y + blockIdx.y*blockDim.y;
#if(IS3D)    
    int k = threadIdx.z + blockIdx.z*blockDim.z;
#else
    int k = 0;
#endif
    double invdx[DIM];
    double xyz[3];
    for (int d = 0; d < DIM; d++) invdx[d] = input.nxb[d]/(input.bounds[2*d+1] - input.bounds[2*d]);
    int dijk[3] = {0};
    int ijk[3] = {0};
    if (i < (input.nxb[0]) && j < (input.nxb[1]) && k < (input.nxb[2]))
    {
        ijk[0] = i;
        ijk[1] = j;
        ijk[2] = k;
        for (int idir = 0; idir < DIM; idir++)
        {
            dijk[idir] = 1;
            for (int d = 0; d < DIM; d++) xyz[d] = input.bounds[2*d] + (ijk[d] + 0.5) / invdx[d];
            
            double stencilData[9*(5+DIM)]; //ie,ke,T,P,rho,u,v,w
            double rhs[2+DIM] = {0.0};
            // fluxes
            double C[2]     = {0.0};
            double M[DIM*2] = {0.0};
            double PGRAD[2] = {0.0};
            double KE[2]    = {0.0};
            double IE[2]    = {0.0};
            double PDIFF[2] = {0.0};

            for (int n = 0; n < input.centOrder + 1; n++)
            {
                for (int v = 3; v < (5+DIM); v++)
                {
                    int ii = i+dijk[0]*(n-stencilWid);
                    int jj = j+dijk[1]*(n-stencilWid);
                    int kk = k+dijk[2]*(n-stencilWid);
                    stencilData[stencilIdx(v,n)] = flow(v-3, ii, jj, kk, lb);
                }
                // T
                stencilData[stencilIdx(2,n)] = stencilData[stencilIdx(3,n)]/(input.Rgas*stencilData[stencilIdx(4,n)]);
                
                // IE = P/(rho*(gamma - 1))
                stencilData[stencilIdx(0,n)] = stencilData[stencilIdx(3,n)]/(stencilData[stencilIdx(4,n)]*(input.gamma - 1.0));
                
                // ke (don't care)
                stencilData[stencilIdx(1,n)] = 0.0;

                // Not needed per se starts
                for (int vel_comp = 0; vel_comp < DIM; vel_comp ++)
                {
                    stencilData[stencilIdx(1,n)] += 0.5*stencilData[stencilIdx(5+vel_comp,n)]*stencilData[stencilIdx(5+vel_comp,n)];
                }
                // Not needed per se ends
            }
            // Mass conservation                              
            for (int l = 1; l <= stencilWid; l++)
            {
                double al = center.c[l-1];
                int jf = stencilWid;
                for (int m = 0; m <= (l-1); m++)
                {
                    C[1] += 2.0*al*fg_QuadSplit(stencilData,jf-m, l,4,5+idir);
                    C[0] += 2.0*al*fg_QuadSplit(stencilData,jf+m,-l,4,5+idir);
                    for (int idir_mom = 0; idir_mom < DIM; idir_mom++)
                    {
                        M[idir_mom      ] += 2.0*al*fg_CubeSplit(stencilData,jf-m, l,4,5+idir,5+idir_mom);
                        M[idir_mom + DIM] += 2.0*al*fg_CubeSplit(stencilData,jf+m,-l,4,5+idir,5+idir_mom);
                    }

                    PGRAD[1] += 2.0*al*f_DivSplit(stencilData,jf-m, l,3);
                    PGRAD[0] += 2.0*al*f_DivSplit(stencilData,jf+m,-l,3);

                    for (int vel_comp = 0;  vel_comp < DIM; vel_comp ++)
                    {
                        KE[1] += 2.0*al*fg_QuadSplit(stencilData,jf-m, l,4,5+idir)*0.5*(stencilData[stencilIdx(5+vel_comp,jf-m)]*stencilData[stencilIdx(5+vel_comp,jf-m+l)]);
                        KE[0] += 2.0*al*fg_QuadSplit(stencilData,jf+m,-l,4,5+idir)*0.5*(stencilData[stencilIdx(5+vel_comp,jf+m)]*stencilData[stencilIdx(5+vel_comp,jf+m-l)]);
                    }

                    IE[1] += 2.0*al*fg_CubeSplit(stencilData,jf-m, l,4,0,5+idir);
                    IE[0] += 2.0*al*fg_CubeSplit(stencilData,jf+m,-l,4,0,5+idir);

                    PDIFF[1] += 2.0*al*fg_DivSplit(stencilData,jf-m, l,5+idir,3);
                    PDIFF[0] += 2.0*al*fg_DivSplit(stencilData,jf+m,-l,5+idir,3);
                }
            }
            
            double pres[4];
            double dens[4];
            double uvel[4];
            double vvel[4];
            double wvel[4];
            double engy[4];
            double rhsExact[5];
            
            pres_mms(pres, xyz[0], xyz[1], xyz[2]);
            dens_mms(dens, xyz[0], xyz[1], xyz[2]);
            uvel_mms(uvel, xyz[0], xyz[1], xyz[2]);
            vvel_mms(vvel, xyz[0], xyz[1], xyz[2]);
            wvel_mms(wvel, xyz[0], xyz[1], xyz[2]);
            
            double invgm1 = 1.0/(input.gamma-1.0);
            engy[0] = pres[0]/(dens[0]*(input.gamma - 1.0)) + 0.5*(sqr(uvel[0]) + sqr(vvel[0]) + IS3D*sqr(wvel[0]));
            engy[1] = (uvel[0]*uvel[1] + vvel[0]*vvel[1] + wvel[0]*wvel[1]) + invgm1*(dens[0]*pres[1]-dens[1]*pres[0])/(sqr(dens[0]));
            engy[2] = (uvel[0]*uvel[2] + vvel[0]*vvel[2] + wvel[0]*wvel[2]) + invgm1*(dens[0]*pres[2]-dens[2]*pres[0])/(sqr(dens[0]));
            engy[3] = (uvel[0]*uvel[3] + vvel[0]*vvel[3] + wvel[0]*wvel[3]) + invgm1*(dens[0]*pres[3]-dens[3]*pres[0])/(sqr(dens[0]));
            
            rhsExact[0] = cont_rhs_mms(pres, dens, uvel, vvel, wvel);
            rhsExact[1] = engy_rhs_mms(pres, dens, uvel, vvel, wvel, engy);
            rhsExact[2] = momx_rhs_mms(pres, dens, uvel, vvel, wvel);
            rhsExact[3] = momy_rhs_mms(pres, dens, uvel, vvel, wvel);
            rhsExact[4] = momz_rhs_mms(pres, dens, uvel, vvel, wvel);

            rhs[0] += invdx[idir]*(C[1] - C[0]);
            rhs[1] += -invdx[idir]*(IE[1] + KE[1] + PDIFF[1] - IE[0] - KE[0] - PDIFF[0]);
            rhs[2+idir] += -invdx[idir]*(PGRAD[1] - PGRAD[0]);
            for (int rhs_vel_comp = 0; rhs_vel_comp < DIM; rhs_vel_comp++)
            {
                rhs[2+rhs_vel_comp] -= invdx[idir]*(M[rhs_vel_comp] - M[rhs_vel_comp+DIM]);
            }
            err(0, i, j, k, lb) = err(0, i, j, k, lb) + (rhs[0] - rhsExact[0]/DIM);
            err(1, i, j, k, lb) = err(1, i, j, k, lb) + (rhs[1] - rhsExact[1]/DIM);
            err(2, i, j, k, lb) = err(2, i, j, k, lb) + (rhs[2] - rhsExact[2]/DIM);
            err(3, i, j, k, lb) = err(3, i, j, k, lb) + (rhs[3] - rhsExact[3]/DIM);
#if(IS3D)
            err(4, i, j, k, lb) = err(4, i, j, k, lb) + (rhs[4] - rhsExact[4]/DIM);
#endif

            dijk[idir] = 0;
        }
    }
}

void GCopy(FlowArr& cTarget, FlowArr& gTarget, size_t size)
{
    CuCheck(hipMemcpy(cTarget.data, gTarget.data, size, hipMemcpyDeviceToHost));
    //need to transpose here too!
}

void ConvGpu(FlowArr& flow, FlowArr& err, const InputClass& input)
{
    dim3 blockConf;
    blockConf.x = BLOCK_SIZEX;
    blockConf.y = BLOCK_SIZEY;
#if(IS3D)
    blockConf.z = BLOCK_SIZEZ;
#endif
    dim3 gridConf;
    int numcells[DIM];
    for (int i = 0; i < DIM; i++) {numcells[i] = input.nxb[i];}
    gridConf.x = (numcells[0] + BLOCK_SIZEX - 1)/BLOCK_SIZEX;
    gridConf.y = (numcells[1] + BLOCK_SIZEY - 1)/BLOCK_SIZEY;
#if(IS3D)
    gridConf.z = (numcells[2] + BLOCK_SIZEZ - 1)/BLOCK_SIZEZ;
#endif

    if (mypenoG==0 && !hasPrintedGp)
    {
        hasPrintedGp = true;
        std::cout << "GP Config:\nblock: " << blockConf.x << " x " << blockConf.y;
        if (IS3D) std::cout << " x " << blockConf.z;
        std::cout << "\ngrid:  " << gridConf.x << " x " << gridConf.y;
        if (IS3D) std::cout << " x " << gridConf.z;
        std::cout << std::endl;
    }
    
    Coef_t center;
    switch (input.centOrder)
    {
        case 2: {center.c[0] = 1.0/2.0; break;}
        case 4: {center.c[0] = 2.0/3.0; center.c[1] = -1.0/12.0; break;}
        case 6: {center.c[0] = 3.0/4.0; center.c[1] = -3.0/20.0; center.c[2] = 1.0/60.0 ;break;}
        case 8: {center.c[0] = 4.0/5.0; center.c[1] = -1.0/5.0 ; center.c[2] = 4.0/105; center.c[3] = -1.0/280.0; break;}
        default: {std::cout << "Bad central scheme order." << std::endl; abort();}
    }
    int stencilWid = input.centOrder/2;
    for (int lb = 0; lb < input.lnblocks; lb++)
    {
        K_Conv<<<gridConf, blockConf>>>(flow, err, input, lb, center, stencilWid);
        CuCheck(hipPeekAtLastError());
    }
    CuCheck(hipDeviceSynchronize());
}

std::string GetGpuKernelDescription(void)
{
    return "Baseline";
}